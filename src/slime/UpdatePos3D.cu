#include "hip/hip_runtime.h"
#include <slime/SlimeSim3D.hpp>
#include <utils/CUDAUtils.h>
#include <glm/gtc/matrix_inverse.hpp>



using namespace slime;


struct EndPos
{
    glm::vec2   Pos;
    glm::vec2   Dir;
    int         TriID;
};

__device__ glm::vec3 D2ToBaryc(glm::vec2 Coords, glm::vec2 V1, glm::vec2 V2, glm::vec2 V3)
{
    float T11 = V1.x - V3.x; // x1 - x3
    float T12 = V2.x - V3.x; // x2 - x3
    float T21 = V1.y - V3.y; // y1 - y3
    float T22 = V2.y - V3.y; // y2 - y3
    float DetT = T11 * T22 - T12 * T21;

    // Lambda1 = ((y2 - y3) * (x - x3) + (x3 - x2) * (y - y3)) / Det(T)
    float L1 = (T22 * (Coords.x - V3.x) - T12 * (Coords.y - V3.y)) / DetT;
    // Lambda2 = ((y3 - y1) * (x - x3) + (x1 - x3) * (y - y3)) / Det(T)
    float L2 = (T11 * (Coords.y - V3.y) - T21 * (Coords.x - V3.x)) / DetT;
    // Lambda3 = 1 - Lambda1 - Lambda2
    return glm::vec3(L1, L2, 1 - L1 - L2);
}

__device__ glm::vec3 D3ToBaryc(glm::vec3 Coords, glm::vec3 P1, glm::vec3 P2, glm::vec3 P3)
{
    glm::mat3 LTo3D(P1, P2, P3);
    return glm::inverse(LTo3D) * Coords;
    // glm::vec3 L(0.0f, 0.0f, 0.0f);

    // // Compute voronoi areas
    // float a = 0.0f;
    // float b = 0.0f;
    // float c = 0.0f;
    // float s = 0.0f;
    // // P1 - P2 - C
    // a = glm::length(P1 - P2);
    // b = glm::length(P2 - Coords);
    // c = glm::length(Coords - P1);
    // s = (a + b + c) / 2.0f;
    // L.z = glm::sqrt(s * (s - a) * (s - b) * (s - c));
    // // P3 - P1 - C
    // a = glm::length(P3 - P1);
    // b = glm::length(P1 - Coords);
    // c = glm::length(Coords - P3);
    // s = (a + b + c) / 2.0f;
    // L.y = glm::sqrt(s * (s - a) * (s - b) * (s - c));

    // // Compute triangle area
    // a = glm::length(P1 - P2);
    // b = glm::length(P2 - P3);
    // c = glm::length(P3 - P1);
    // s = (a + b + c) / 2.0f;
    // s = glm::sqrt(s * (s - a) * (s - b) * (s - c));
    
    // // Return
    // L /= s;
    // L.x = 1 - L.z - L.y;
    // return L;
}

__device__ glm::vec2 BarycToD2(glm::vec3 Coords, glm::vec2 V1, glm::vec2 V2, glm::vec2 V3)
{
    return Coords.x * V1 + Coords.y * V2 + Coords.z * V3;
}

__device__ glm::vec3 BarycToD3(glm::vec3 Coords, glm::vec3 V1, glm::vec3 V2, glm::vec3 V3)
{
    return Coords.x * V1 + Coords.y * V2 + Coords.z * V3;
}

__device__ glm::vec3 TriNorm(glm::vec3 P1, glm::vec3 P2, glm::vec3 P3)
{
    // Get edges
    glm::vec3 E12 = P2 - P1;
    glm::vec3 E23 = P3 - P2;
    glm::vec3 E31 = P1 - P3;

    // Compute normal
    glm::vec3 N = glm::cross(E12, -E31);
    N = N + glm::cross(E23, -E12);
    N = N + glm::cross(E31, -E23);
    N = glm::normalize(N);

    return N;
}

__device__ glm::vec4 ProjOnTri(glm::vec3 P, glm::vec3 P1, glm::vec3 P2, glm::vec3 P3)
{
    // Get edges
    glm::vec3 E12 = P2 - P1;
    glm::vec3 E23 = P3 - P2;
    glm::vec3 E31 = P1 - P3;

    // Compute normal
    glm::vec3 N = glm::cross(E12, -E31);
    N = N + glm::cross(E23, -E12);
    N = N + glm::cross(E31, -E23);
    N = glm::normalize(N);

    // Compute centre
    glm::vec3 C = (P1 + P2 + P3) / 3.0f;

    // Project CP onto normal
    float Proj = glm::dot(P - C, N);

    // Remove projection from P
    return glm::vec4(P - Proj * N, Proj);
}


__device__ EndPos CalcEndPositionStep(glm::vec2 Pos, glm::vec2 Move, int TriID, mesh::Triangle* Tris, mesh::Vertex* Verts, glm::ivec3* T2T, float* UV3D)
{
    EndPos ep;
    ep.Pos = Pos;
    ep.Dir = Move;
    ep.TriID = TriID;

    // Get the vertex indices of the triangles
    glm::ivec3 TV = Tris[TriID].Verts;

    // Get the vertices
    glm::vec2 UV1 = Verts[TV[0]].TexUV;
    glm::vec2 UV2 = Verts[TV[1]].TexUV;
    glm::vec2 UV3 = Verts[TV[2]].TexUV;

    // Convert to barycentric
    glm::vec3 L = D2ToBaryc(Pos + Move / UV3D[TriID], UV1, UV2, UV3);

    // Non-negative coordinates = point inside
    if (L.x >= 0 && L.y >= 0 && L.z >= 0)
    {
        ep.Pos += Move / UV3D[TriID];
        return ep;
    }

    // Find the edge we crossed while going out. It is the negative coordinate
    int CrossEdge = 0;
    for (int i = 0; i < 3; ++i)
    {
        if (L[i] < 0)
        {
            CrossEdge = i;
            break;
        }
    }

    // Get the triangle where we end to
    int AdjTri = T2T[TriID][CrossEdge];
    // If we are at boundary, don't move and go back
    if (AdjTri < 0)
    {
        ep.Dir = -Move;
        return;
    }

    // Get 3D position of point w.r.t. current triangle
    glm::vec3 V1 = Verts[TV[0]].Position;
    glm::vec3 V2 = Verts[TV[1]].Position;
    glm::vec3 V3 = Verts[TV[2]].Position;
    glm::vec3 P3D = BarycToD3(L, V1, V2, V3);
    glm::vec3 Orig3D = BarycToD3(D2ToBaryc(Pos, UV1, UV2, UV3), V1, V2, V3);

    // Project on the adjacent triangle
    TV = Tris[AdjTri].Verts;
    V1 = Verts[TV[0]].Position;
    V2 = Verts[TV[1]].Position;
    V3 = Verts[TV[2]].Position;
    glm::vec4 Proj = ProjOnTri(P3D, V1, V2, V3);
    glm::vec3 NewP3D(Proj.x, Proj.y, Proj.z);
    Proj = ProjOnTri(P3D + 1.0f * (P3D - Orig3D), V1, V2, V3);
    glm::vec3 Far3D(Proj.x, Proj.y, Proj.z);

    // Get barycentric coordinates, and adjust
    L = D3ToBaryc(NewP3D, V1, V2, V3);
    L /= (L.x + L.y + L.z);

    // Back to 2D
    UV1 = Verts[TV[0]].TexUV;
    UV2 = Verts[TV[1]].TexUV;
    UV3 = Verts[TV[2]].TexUV;
    ep.Pos = BarycToD2(L, UV1, UV2, UV3);

    // Pick 2D of far point
    L = D3ToBaryc(Far3D, V1, V2, V3);
    L /= (L.x + L.y + L.z);
    ep.Dir = BarycToD2(L, UV1, UV2, UV3);

    // Get new triangle ID
    ep.TriID = AdjTri;
    ep.Dir -= ep.Pos;
    ep.Dir = glm::length(Move) * glm::normalize(ep.Dir);
    return ep;
}

__device__ EndPos CalcEndPosition(glm::vec2 Pos, glm::vec2 Move, int TriID, mesh::Triangle* Tris, mesh::Vertex* Verts, glm::ivec3* T2T, float* UV3D, SimulationParameters Params)
{
    // Get the number of steps and scale the movement
    int NumSteps = (int)glm::ceil(glm::length(Move) / Params.MoveStep);
    Move = Params.MoveStep * glm::normalize(Move);
    // Apply steps iteratively
    EndPos ep = { Pos, Move, TriID };
    float TotMove = 0.0f;
    for (int i = 0; i < NumSteps; ++i)
        ep = CalcEndPositionStep(ep.Pos, ep.Dir, ep.TriID, Tris, Verts, T2T, UV3D);
    return ep;
}

__device__ float Sense(glm::vec2 Centre, int SpeciesID, float* TrailMap, unsigned char* Obstacle, int Width, int Height, bool IsObstacle, SimulationParameters Params)
{
    glm::ivec2 Coords(int(Centre.x * (Width - 1)), int(Centre.y * (Height - 1)));
    float Sum = 0.0f;
    for (int dx = -Params.SensorRadius; dx <= Params.SensorRadius; ++dx)
    {
        int X = glm::clamp(Coords.x + dx, 0, Width - 1);
        for (int dy = -Params.SensorRadius; dy <= Params.SensorRadius; ++dy)
        {
            int Y = glm::clamp(Coords.y + dy, 0, Height - 1);
            int Idx = Y * Width + X;
            for (int k = 0; k < Params.NumSpecies; ++k)
            {
                float Sign = (k == SpeciesID) ? 1.0f : -1.0f;
                Sum += Sign * TrailMap[Idx * Params.NumSpecies + k];
            }
            if (Obstacle != NULL && IsObstacle)
            {
                float ObstacleValue = Obstacle[Idx] / 255.0f;
                Sum -= (TrailMap[Idx * Params.NumSpecies + SpeciesID] + 1) * ObstacleValue * 10.0f;
            }
        }
    }
    return Sum;
}

__device__ Agent NextDir(Agent A, float* TrailMap, unsigned char* Obstacle, int Width, int Height, bool IsObstacle,
                         mesh::Vertex* Verts, mesh::Triangle* Tris, glm::ivec3* T2T, float* UV3D,
                         SimulationParameters Params)
{
    // Get direction
    glm::vec2 Dir(glm::cos(A.Angle), glm::sin(A.Angle));

    // Look forward
    EndPos SensorFwd = CalcEndPosition(A.Pos, Dir * Params.VisionDist, A.TriID, Tris, Verts, T2T, UV3D, Params);
    float Fwd = Sense(SensorFwd.Pos, A.SpeciesID, TrailMap, Obstacle, Width, Height, IsObstacle, Params);

    // Look right
    Dir.x = glm::cos(A.Angle + glm::radians(Params.VisionAngle));
    Dir.y = glm::sin(A.Angle + glm::radians(Params.VisionAngle));
    EndPos SensorRight = CalcEndPosition(A.Pos, Dir * Params.VisionDist, A.TriID, Tris, Verts, T2T, UV3D, Params);
    float Right = Sense(SensorRight.Pos, A.SpeciesID, TrailMap, Obstacle, Width, Height, IsObstacle, Params);

    // Look left
    Dir.x = glm::cos(A.Angle - glm::radians(Params.VisionAngle));
    Dir.y = glm::sin(A.Angle - glm::radians(Params.VisionAngle));
    EndPos SensorLeft = CalcEndPosition(A.Pos, Dir * Params.VisionDist, A.TriID, Tris, Verts, T2T, UV3D, Params);
    float Left = Sense(SensorLeft.Pos, A.SpeciesID, TrailMap, Obstacle, Width, Height, IsObstacle, Params);


    // If greater concentration of pheromone is forward, go ahead
    if (Fwd > Right && Fwd > Left)
        return A;
    
    // Determine direction, and add a bit of randomness
    A.RandState = RandHash(A.RandState);
    float RandSteer = ScaleTo01(A.RandState);
    float Turn = Params.TurnSpeed * Params.DeltaTime * 3.14159265;

    // If concentration forward is very small, act randomly
    // if (Fwd < Left && Fwd < Right)
    //     A.Angle += (RandSteer - 0.5f) * 2 * Turn;
    // If left is greater, go left
    if (Left > Right)
        A.Angle -= RandSteer * Turn;
    // If right is greater, go right
    else
        A.Angle += RandSteer * Turn;

    return A;
}


__global__ void UpdatePositionsKernel(Agent* Agents, float* TrailMap, unsigned char* Obstacle, int Width, int Height, 
                                      bool IsObstacle, SimulationParameters Params,
                                      mesh::Triangle* Tris, mesh::Vertex* Verts, glm::ivec3* T2T, float* UV3D)
{
    int AgentID = blockDim.x * blockIdx.x + threadIdx.x;
    if (AgentID >= Params.NumAgents)
        return;

    // Get the agent and determine direction
    Agent A = Agents[AgentID];
    A = NextDir(A, TrailMap, Obstacle, Width, Height, IsObstacle, Verts, Tris, T2T, UV3D, Params);

    // Move the agent
    glm::vec2 Dir(glm::cos(A.Angle), glm::sin(A.Angle));
    EndPos ep = CalcEndPosition(A.Pos, Dir * Params.MoveSpeed * Params.DeltaTime, A.TriID, Tris, Verts, T2T, UV3D, Params);
    A.Pos = ep.Pos;
    A.Angle = glm::atan(ep.Dir.y, ep.Dir.x);
    A.TriID = ep.TriID;
    Agents[AgentID] = A;

    // Get the agent position in trail map
    int X = (int)(A.Pos.x * (Width - 1));
    int Y = (int)(A.Pos.y * (Height - 1));
    X = glm::clamp(X, 0, Width - 1);
    Y = glm::clamp(Y, 0, Height - 1);
    int Idx = Y * Width + X;
    TrailMap[Idx * Params.NumSpecies + A.SpeciesID] = 1.0f;
}




void slime::SlimeSim3D::LaunchUpdatePositionsKernel()
{
    dim3 bSize(1024);
    dim3 gSize((Params.NumAgents + bSize.x - 1) / bSize.x);
    UpdatePositionsKernel<<<gSize, bSize>>>(dAgents, dTrailMap, dObstacle, TrailMapTex.Width, TrailMapTex.Height, IsObstacle, Params, dTris, dVerts, dT2T, UVTo3D);
    cudaErrorCheck(hipDeviceSynchronize());
}